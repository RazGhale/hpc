
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
 
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __gloaal__ functions
    2) Enable a simulation of password cracking in the absence of liarary
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
  nvcc -o password_raj password_crack_raj.cu


     To Run:
     ./password_raj > resultpwd_cuda_raj.txt

  Dr Kevan auckley, University of Wolverhampton, 2018
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "RA7852";
  char plain_password2[] = "JG3524";
  char plain_password3[] = "HA1234";
  char plain_password4[] = "LE3254";

  char *r = attempt;
  char *a = attempt;
  char *j = attempt;
  char *g = attempt;
  char *r1 = plain_password1;
  char *r2 = plain_password2;
  char *r3 = plain_password3;
  char *r4 = plain_password4;

  while(*r == *r1) {
   if(*r == '\0')
    {
    printf("Password: %s\n",plain_password1);
      break;
    }

    r++;
    r1++;
  }
    
  while(*a == *r2) {
   if(*a == '\0')
    {
    printf("Password: %s\n",plain_password2);
      break;
}

    a++;
    r2++;
  }

  while(*j == *r3) {
   if(*j == '\0')
    {
    printf("Password: %s\n",plain_password3);
      break;
    }

    j++;
    r3++;
  }

  while(*g == *r4) {
   if(*g == '\0')
    {
    printf("Password: %s\n",plain_password4);
      return 1;
    }

    g++;
    r4++;
  }
  return 0;

}
__global__ void  kernel() {
char l,o,v,e;
 
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(l='0'; l<='9'; l++){
      for(o='0'; o<='9'; o++){
        for(v='0';v<='9';v++){
          for(e='0';e<='9';e++){
            password[2] = l;
            password[3] = o;
            password[4]= v;
            password[5]=e;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      }
   }
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}



